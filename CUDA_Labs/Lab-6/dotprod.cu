#include "hip/hip_runtime.h"
% % cu
#include <bits/stdc++.h>
	using namespace std;
#define N 1500
#define M 1024

__global__ void dot_product(float *a, float *b, float *c)
{
	__shared__ float temp[M];

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		temp[threadIdx.x] = a[index] * b[index];
		__syncthreads();
		if (threadIdx.x == 0)
		{
			float sum = 0.0;
			if (blockIdx.x < N / blockDim.x)
			{
				for (int i = 0; i < (int)blockDim.x; i++)
					sum += temp[i];
			}
			else
			{
				for (int i = 0; i < N % blockDim.x; i++)
					sum += temp[i];
			}
			atomicAdd(c, sum);
		}
	}
}

int main()
{
	srand(time(0));
	int blocks[] = {1, 1, 1, 1, 1, 1, 1, 10, 20, 30, 40, 50, M / 8, M / 4, M / 2, M, M, M, M, M};
	int threads[] = {1, 10, 20, 30, 40, 50, M, 10, 10, 10, 10, 10, M, M, M, M / 8, M / 4, M / 2, M};
	float a[N], b[N], c[N];
	float *d_a, *d_b, *d_c;
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 0.250;
		b[i] = i + 0.248;
	}

	hipMalloc((void **)&d_a, N * sizeof(float));
	hipMalloc((void **)&d_b, N * sizeof(float));
	hipMalloc((void **)&d_c, N * sizeof(float));

	hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

	for (int k = 0; k < 19; k++)
	{
		float elapsed = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		dot_product<<<blocks[k], threads[k]>>>(d_a, d_b, d_c);

		hipError_t err = hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
			cout << "CUDA Error copying to Host: " << hipGetErrorString(err);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsed, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);
		printf("Blocks = %4d and Threads per Block = %4d Time = %.5f\n", blocks[k], threads[k], elapsed);
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}