#include "hip/hip_runtime.h"
% % cu
#include <bits/stdc++.h>
	using namespace std;
#define N 7
#define M 1024

__global__ void matadd(double a[][N], double b[][N], double c[][N])
{
	int id = gridDim.x * blockIdx.y + blockIdx.x;
	if (id < N)
	{
		for (int i = 0; i < N; i++)
			c[id][i] = a[id][i] + b[id][i];
	}
}

int main()
{
	srand(time(0));
	int blocks[] = {1, 1, 1, 1, 1, 1, 1, 10, 20, 30, 40, 50, M / 8, M / 4, M / 2, M, M, M, M, M};
	int threads[] = {1, 10, 20, 30, 40, 50, M, 10, 10, 10, 10, 10, M, M, M, M / 8, M / 4, M / 2, M};
	double a[N][N], b[N][N], c[N][N] = {{0}};
	double(*d_a)[N], (*d_b)[N], (*d_c)[N];
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			a[i][j] = i + j + 0.250;
			b[i][j] = i + j + 0.248;
		}
	}
	hipMalloc((void **)&d_a, N * N * sizeof(double));
	hipMalloc((void **)&d_b, N * N * sizeof(double));
	hipMalloc((void **)&d_c, N * N * sizeof(double));

	hipMemcpy(d_a, a, N * N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N * N * sizeof(double), hipMemcpyHostToDevice);
	dim3 grid(N, N);

	for (int k = 0; k < 19; k++)
	{
		float elapsed = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		matadd<<<blocks[k], threads[k]>>>(d_a, d_b, d_c);

		hipError_t err = hipMemcpy(c, d_c, N * N * sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
			cout << "CUDA Error copying to Host: " << hipGetErrorString(err);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsed, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);
		printf("Blocks = %4d and Threads per Block = %4d Time = %.5f\n", blocks[k], threads[k], elapsed);
	}

	printf("\nMatrix A:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)

			cout << a[i][j] << "\t";

		cout << endl;
	}
	printf("\nMatrix B:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			cout << b[i][j] << "\t";
		cout << endl;
	}
	printf("\nSum :\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			cout << c[i][j] << "\t";
		cout << endl;
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}