#include "hip/hip_runtime.h"
% % cu
#include <bits/stdc++.h>
	using namespace std;
#define N 25
#define M 1024

__global__ void vector_mul(double *a, double *b, double *c)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < N)
		c[id] = a[id] * b[id];
}

int main()
{
	srand(time(0));
	int blocks[] = {1, 1, 1, 1, 1, 1, 1, 10, 20, 30, 40, 50, M / 2, M / 4, M / 8, M, M, M, M, M};
	int threads[] = {1, 10, 20, 30, 40, 50, M, 10, 10, 10, 10, 10, M, M, M, M / 2, M / 4, M / 8, M};
	double a[N], b[N], c[N];
	double *d_a, *d_b, *d_c;
	double size = N * sizeof(double);

	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	for (int i = 0; i < N; i++)
	{
		a[i] = rand() % 100 + i + 0.250;
		b[i] = rand() % 100 * i + 0.248;
	}

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	for (int k = 0; k < 19; k++)
	{
		float elapsed = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0);
		vector_mul<<<blocks[k], threads[k]>>>(d_a, d_b, d_c);

		// Copy result back to host
		hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
			cout << "CUDA Error copying to Host :" << hipGetErrorString(err) << endl;

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsed, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);

		printf("Blocks = %4d and Threads per Block = %4d Time = %.5f\n", blocks[k], threads[k], elapsed);
	}
	cout << "\nProduct of Vectors " << endl;
	for (int i = 0; i < N; i++)
		cout << a[i] << " * " << b[i] << " = " << c[i] << endl;

	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}