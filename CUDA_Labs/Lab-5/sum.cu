#include "hip/hip_runtime.h"
% % cu
#include <bits/stdc++.h>
	using namespace std;
#define N 1500
#define M 1024

__global__ void N_sum(float *a, float *b)
{
	__shared__ float temp[M];
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
	{
		temp[threadIdx.x] = a[index];
		__syncthreads();
		if (threadIdx.x == 0)
		{
			float sum = 0;
			for (int i = 0; i < M; i++)
				sum += temp[i];
			atomicAdd(b, sum);
		}
	}
}

int main()
{
	srand(time(0));
	int blocks[] = {1, 1, 1, 1, 1, 1, 1, 10, 20, 30, 40, 50, M / 8, M / 4, M / 2, M, M, M, M, M};
	int threads[] = {1, 10, 20, 30, 40, 50, M, 10, 10, 10, 10, 10, M, M, M, M / 8, M / 4, M / 2, M};
	float a[N], b[N] = {0};
	float *d_a, *d_b;
	for (int i = 0; i < N; i++)
		a[i] = i + 1.2564;
	hipMalloc((void **)&d_a, N * sizeof(float));
	hipMalloc((void **)&d_b, N * sizeof(float));

	hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

	for (int k = 0; k < 19; k++)
	{
		float elapsed = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		N_sum<<<blocks[k], threads[k]>>>(d_a, d_b);

		hipError_t err = hipMemcpy(b, d_b, N * sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
			cout << "CUDA Error copying to Host: " << hipGetErrorString(err);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsed, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);
		printf("Blocks = %4d and Threads per Block = %4d Time = %.5f\n", blocks[k], threads[k], elapsed);
	}

	printf("\nNumbers : ");
	for (int i = 0; i < N; i++)
		cout << a[i] << "  ";

	printf("\nSum     : ");
	cout << b[0] / 19.0 << endl;
	hipFree(d_a);
	hipFree(d_b);

	return 0;
}